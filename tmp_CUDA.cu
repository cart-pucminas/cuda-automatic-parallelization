#include "hip/hip_runtime.h"
// #include <omp.h>
// #include <stdio.h>
// #include <stdlib.h>


size_t number_of_blocks = 1000;

size_t threads_per_block = 1000;



//Chamada da função GPUFuncion
__global__ void GPUFunction(double nCuda , valor , double valor ){

int idx = blockIdx.x * blockDim.x + threadIdx.x;

int totalThreads = gridDim.x * blockDim.x;

for (double idxCuda = idx; idxCuda < nCuda ; idxCuda += totalThreads){

//parte que será paralelizada
valor = valor + 4.0/(1.0 + ((idx + 0.5)*(1.0/(double)2147480000))*((idx + 0.5)*(1.0/(double)2147480000)));
}

}



int main(int argc, char** argv){

double valor = 0;
// double passo;
// passo = 1.0/(double)2147480000;

GPUFunction<<<number_of_blocks, threads_per_block>>>(2147480000, count, valor);

hipDeviceSynchronize();




valor = valor*(1.0/(double)2147480000);

printf("%f\n", valor);
return 0;
}
