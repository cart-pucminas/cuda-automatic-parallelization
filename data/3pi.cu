
#include <hip/hip_runtime.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>



size_t number_of_blocks = 1000;
size_t threads_per_block = 1000;

__global__ void GPUFunction(double n, double passo, double *pi){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int totalThreads = gridDim.x * blockDim.x;

    for (double i = idx; i < n ; i += totalThreads){    
        //Parte que será alterada
        atomicAdd(pi, (4.0/(1.0 + ((i + 0.5)*passo)*(i + 0.5)*passo)));
    }
   
    
}

int main(int argc, char** argv){
    long long num_passos = 10000000000;
    double passo;

    double *pi;
    int n = 1; 
    hipMallocManaged(&pi, n * sizeof(double));
    pi[0] = 0;

    passo = 1.0/(double)num_passos;    

    

    GPUFunction<<<number_of_blocks, threads_per_block>>>(num_passos, passo, pi);
    hipDeviceSynchronize();

    printf("O valor de PI é: %f\n", (pi[0]*passo));
    return 0;
}


// 4999800002.000000
