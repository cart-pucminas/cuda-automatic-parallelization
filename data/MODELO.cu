#include "hip/hip_runtime.h"
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>



// FALTA ALTERAR:
// Code precisa criar código na estrutura CUDA e executar esse código
// As alterações são mais complexas do que somente alterar um pedaço do pragma
// Os experimentos serão realizados com 3 algoritmos: super simples, preenchimento de vetor e cálculo do PI
// Preciso explicar para o Humberto os trabalhos futuros e as dificuldades encontradas

// COMPORTAMENTO DO ALGORITMO
// o NeurCUDA precisa criar a estrutura cuda e testar
// Trocar as variáveis para compartilhada e privada e testar cada troca
// Criar reduction em cada variável e testar cada troca

// Variável compartilhada em CUDA: __shared__ int y;
// Variável privada em CUDA: normal já é privada


// 1) Realiza a análise TREE no código e define as variáveis FEITO
// 2) Encontra o neuromp no código e o for que deve-se paraleliza. FEITO
// 3) Define as linhas que pertencem ao for e retira do código, levando para uma função separada, arrumando as declarações de variáveis. FEITO
// 4) Constrói a Função GPU e a chamada dela com as variáveis que ela usa. 
// 5) Contruir listas com os dados das variáveis FEITO
// 5) Testa (usando aprendizagem por reforço) cada variáveis pertencente como retorno, alocando memória e afins. (lembrar de olhar se é vetor e afins, alocando espaço corretamente)
// 6) Testa (usando aprendizagem por reforço) cada variáveis pertencente como retorno, alocando memória e afins e usando o atomicSum nesse variável para a operação correspondente.

//APRENDIZAGEM POR REFORÇO
// 1) Testes com ponteiro e alocando memória em cada variável
// 2) Testes colocando o atomicSum em cada variável

size_t number_of_blocks = 1000;
size_t threads_per_block = 1000;

__global__ void GPUFunction(double *result, OUTRAS, double n){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int totalThreads = gridDim.x * blockDim.x;

    for (double i = idx; i < n ; i += totalThreads){    
        //Parte que será alterada
       
    }
}

int main(int argc, char** argv){
    long long num_passos = 10000000000;

    double *result; //variável de escrita
    int n = 1; //se vetor tiver algum tamanho
    hipMallocManaged(&result, n * sizeof(double));

    GPUFunction<<<number_of_blocks, threads_per_block>>>(result, OUTRAS);
    hipDeviceSynchronize();

    printf("resultado: %f\n", (resultado[0]));
    return 0;
}


