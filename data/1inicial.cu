

#include <hip/hip_runtime.h>
#include <stdio.h>

size_t number_of_blocks = 1000;
size_t threads_per_block = 1000;

__global__ void GPUFunction(double n, int *valor){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int totalThreads = gridDim.x * blockDim.x;
    for (int i = idx; i < n ; i += totalThreads){

        
        //Parte que será alterada
        for (int j=1; j < n; j++){
            valor[0] = 2;
        } 
        

    }
}

int main(int argc, char** argv){
    double num_passos = 110000;

    //variável que é a saída do código int valor trasformada em:
    int n = 1;
    int *valor;
    size_t size = n * sizeof(int);
    hipMallocManaged(&valor, size);

    GPUFunction<<<number_of_blocks, threads_per_block>>>(num_passos, valor);
    hipDeviceSynchronize();

    printf("O valor é: %d\n", valor[0]);

    return 0;
}
