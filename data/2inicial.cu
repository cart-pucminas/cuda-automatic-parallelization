
#include <hip/hip_runtime.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>


// __global__ void GPUFunction(double n, int *valor){
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     int totalThreads = gridDim.x * blockDim.x;

    
//     extern __shared__ int partials[50000];
//     partials[idx] = 0;

    
//     for (int i = idx; i < n ; i += totalThreads){    
//         //Parte que será alterada
//         for (int j=1; j < n; j++){
//             partials[idx] += 2;
            
//             // valor[0] = valor[0]+2;
//         } 
//     }
    

//     int i = idx / 2;
//     while (i != 0) {
//         /* if we are part of this round */
//         if (idx < i) {
//             /* add the one to our right by i places into this one */
//             partials[idx] += partials[idx + i];
//         }

//         /* cut i in half */
//         i /= 2;
//     }

//     if (idx == 0) {
//         *valor = partials[0];
//     }


        

// }


size_t number_of_blocks = 1000;
size_t threads_per_block = 1000;

__global__ void GPUFunction(double n, int *valor){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int totalThreads = gridDim.x * blockDim.x;

    for (int i = idx; i < n ; i += totalThreads){    
        //Parte que será alterada
        for (int j=0; j < n; j++){
            atomicAdd(valor, 2);
            // valor[0] = valor[0]+2;
        } 
    }
   
    
}

int main(int argc, char** argv){
    double num_passos = 50000;

    //variável que é a saída do código int valor trasformada em:
    int n = 1;
    int *valor;
    size_t size = n * sizeof(int);
    hipMallocManaged(&valor, size);

    GPUFunction<<<number_of_blocks, threads_per_block>>>(num_passos, valor);
    hipDeviceSynchronize();

    printf("O valor é: %d\n", valor[0]);

    return 0;
}


// 4999800002.000000
