#include "hip/hip_runtime.h"


size_t number_of_blocks = 1000;

size_t threads_per_block = 1000;



//Chamada da função GPUFuncion
__global__ void GPUFunction(double nCuda , int icount , double num_passos , double pi){

int idx = blockIdx.x * blockDim.x + threadIdx.x;

int totalThreads = gridDim.x * blockDim.x;

for (double idxCuda = idx; idxCuda < nCuda ; idxCuda += totalThreads){

//parte que será paralelizada
pi += 4.0/(1.0 + ((i + 0.5)*passo)*((i + 0.5)*passo));
}

}



int main(int argc, char** argv){
double num_passos = 10000000000;
double pi=0;
int icount = 0;
double passo = 1.0/(double)num_passos;

GPUFunction<<<number_of_blocks, threads_per_block>>>(1000, icount, passo, pi,);

hipDeviceSynchronize();




pi = pi*passo;

printf("O valor de PI é: %f\n", pi);
return 0;
}
